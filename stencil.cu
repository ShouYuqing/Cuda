#include "hip/hip_runtime.h"
// CUDA implementation of 1d_stencil
// compile: $ nvcc -o t280 t280.cu
//          $ cuda-memcheck ./t280
//
#define N 30
#define THREADS_PER_BLOCK 10
#define BLOCK_SIZE THREADS_PER_BLOCK
#define RADIUS 3

#include <stdio.h>

void random_ints(int *var, int n) // Attribue une valeur à toutes le composantes des variables
{
    int i;
    for (i = 0; i < n; i++)
        var[i] = 1;
}

// 1d_stencil function runnned on multi_block multi_thread
__global__ void stencil_1d(int *in, int *out) {
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];  // shared memory in block 

    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) {
        temp[lindex - RADIUS] = (gindex >= RADIUS)?in[gindex - RADIUS]:0; 
        temp[lindex + BLOCK_SIZE] = ((gindex + BLOCK_SIZE)<N)?in[gindex + BLOCK_SIZE]:0; 

    __syncthreads(); 
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS ; offset++)
        result += temp[lindex + offset];

    out[gindex] = result;
}

int main()
{
    int size = N * sizeof(int);

    int *in, *out; 
    int *d_in, *d_out;  

    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);

    in = (int*)malloc(size); random_ints(in, N);
    out = (int*)malloc(size);
    
    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice); // copy memory from CPU to GPU
    hipMemcpy(d_out, out, size, hipMemcpyHostToDevice);
  
    stencil_1d <<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_in, d_out);// kernel function
  
    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost); // copy the result from GPU to CPU

    for(int i=0; i<N; i++)
        printf("%i ---i=%i \n", out[i], i);

    free(in); free(out);
  
    hipFree(d_in); hipFree(d_out);

    return 0;
}