#include "hip/hip_runtime.h"
//Cuda hello world
#include<stdio.h>
#define N 10
#define THREADS_PER_BLOCK 1
#define BLOCK_SIZE THREADS_PER_BLOCK

// calculation of loss
__global__ void cal_loss(int *error, int *label, int N) {
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int totalPos = blockDim.x * gridDim.x;

	for (int idx = N * pos / totalPos; idx < N * (pos+1) / totalPos; ++idx) { 
		err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]); // calculation of error
	}
	return 0;
}

int main()
{
	int size = 10 * sizeof(float);
	float *label = [0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 1.0f, 0.0f, 0.0f];
    float *d_label;
    hipMalloc(&d_label, sizeof(float) * size);
    hipMemcpy(d_label, label, size, size, hipMemcpyHostToDevice);




    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);

    in = (int*)malloc(size); random_ints(in, N);
    out = (int*)malloc(size);
    
    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice); // copy memory from CPU to GPU
    hipMemcpy(d_out, out, size, hipMemcpyHostToDevice);
  
    stencil_1d <<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_in, d_out);// kernel function
  
    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost); // copy the result from GPU to CPU

    for(int i=0; i<N; i++)
        printf("%i ---i=%i \n", out[i], i);

    free(in); free(out);
  
    hipFree(d_in); hipFree(d_out);

    return 0;
}

