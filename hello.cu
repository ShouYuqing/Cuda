//Cuda hello world

#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 60
#define THREADS_PER_BLOCK 6
#define BLOCK_SIZE THREADS_PER_BLOCK

// calculation of loss
__global__ void cal_loss(float *err, float *label, int n){

	printf("threadIdx:(%d) blockIdx:(%d)\n "
	, threadIdx.x, blockIdx.x);
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int totalPos = blockDim.x * gridDim.x;

	//for (int idx = N * pos / totalPos; idx < N * (pos+1) / totalPos; ++idx) { 
	//	err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]); // calculation of error
	//}
}

int main()
{
	// host data
	int size = 10 * sizeof(float);
	//float *label = [0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 1.0f, 0.0f, 0.0f];
	float *label;
	float *err;
    err = (float*)malloc(size);
    label = (float*)malloc(size);

    // copy data to device
    float *d_label, *d_err;
    hipMalloc(&d_label, size);
    hipMalloc(&d_err, sizeof(float) * 10);
    hipMemcpy(d_label, label, size, hipMemcpyHostToDevice);
    hipMemcpy(d_err, err, size, hipMemcpyHostToDevice);
 
    cal_loss <<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_err, d_label, 10);// kernel function
  
    //cudaMemcpy(out, d_out, size, cudaMemcpyDeviceToHost); // copy the result from GPU to CPU

    //for(int i=0; i<N; i++)
    //    printf("%i ---i=%i \n", out[i], i);

    free(label); free(err);
  
    hipFree(d_label); hipFree(d_err);

    return 0;
}

