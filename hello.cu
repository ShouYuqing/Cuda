#include "hip/hip_runtime.h"
//Cuda hello world
#include<stdio.h>
#define N 10
#define THREADS_PER_BLOCK 1
#define BLOCK_SIZE THREADS_PER_BLOCK

// calculation of loss
__global__ void cal_loss(int *err, int *label, int N) {

	printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) "
	"gridDim:(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z,
	blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
	gridDim.x,gridDim.y,gridDim.z);
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int totalPos = blockDim.x * gridDim.x;

	//for (int idx = N * pos / totalPos; idx < N * (pos+1) / totalPos; ++idx) { 
	//	err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]); // calculation of error
	//}
	return 0;
}

int main()
{
	// host data
	int size = 10 * sizeof(float);
	//float *label = [0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 1.0f, 0.0f, 0.0f];
	float *label;
	float *err;
    err = (float*)malloc(size);
    label = (float*)malloc(size);

    // copy data to device
    float *d_label, *d_err;
    hipMalloc(&d_label, size);
    hipMalloc(&d_err, sizeof(float) * 10);
    hipMemcpy(d_label, label, size, hipMemcpyHostToDevice);
    hipMemcpy(d_err, err, size, hipMemcpyHostToDevice);
 
    cal_loss <<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_err, d_label, 10);// kernel function
  
    //hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost); // copy the result from GPU to CPU

    //for(int i=0; i<N; i++)
    //    printf("%i ---i=%i \n", out[i], i);

    free(label); free(err);
  
    hipFree(d_label); hipFree(d_err);

    return 0;
}

