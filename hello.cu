//Cuda hello world

#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 10
#define THREADS_PER_BLOCK 1
#define BLOCK_SIZE THREADS_PER_BLOCK

// calculation of loss
__global__ void cal_loss(float *err, float *label, int n){

	printf("threadIdx:(%d) blockIdx:(%d)\n "
	, threadIdx.x, blockIdx.x);
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int totalPos = blockDim.x * gridDim.x;

	//for (int idx = N * pos / totalPos; idx < N * (pos+1) / totalPos; ++idx) { 
	//	err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]); // calculation of error
	//}
}

void init_float(float * arr, int size){
	for (int i = 0; i < size; ++i)
	{
		if(i == 3)
			arr[i] = 1;
		else
			arr[i] = 0;
	}
}


int main()
{
	// host data
	int size = 10 * sizeof(float);
	float *label = (float*)malloc(size);
	init_float(label, 10);
	float *err;
    err = (float*)malloc(size);
    label = (float*)malloc(size);

    // copy data to device
    float *d_label, *d_err;
    hipMalloc(&d_label, size);
    hipMalloc(&d_err, sizeof(float) * 10);
    hipMemcpy(d_label, label, size, hipMemcpyHostToDevice);
    hipMemcpy(d_err, err, size, hipMemcpyHostToDevice);
 
    cal_loss <<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_err, d_label, 10);// kernel function
  
    //cudaMemcpy(out, d_out, size, cudaMemcpyDeviceToHost); // copy the result from GPU to CPU

    free(label); free(err);
  
    hipFree(d_label); hipFree(d_err);

    return 0;
}

